#include "hip/hip_runtime.h"
// %%cu

/*** V_2 ***/

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>


__global__ void v_2(int* read, int* write, int n, int moments_per_thread_sqrt);

int main() {

    int nV[] = { 1024, 4000, 8000, 12000, 16000 };
    int kV[] = { 10, 28, 30, 32, 35 };
    int bV[] = { 2, 3, 4, 5, 6, 7, 8};

    double myTime = 0.0;
    struct timeval start,end;

    int threads_per_block_sqrt = 32;

    for(int index=0; index < 5; ++index) {

       for(int b_index=0; b_index < 7; ++b_index) {

            int n = nV[index];
            int moments_per_thread_sqrt = bV[b_index];
            int num_blocks_sqrt = n / (threads_per_block_sqrt * moments_per_thread_sqrt);
            int size = n * n;
            int k = kV[index];

            printf("\nV2: n=%d, k=%d, [blocks=%d, threads/block=%d, moments/thread=%d, all squared]\n\n",
                   n, k, num_blocks_sqrt, threads_per_block_sqrt, moments_per_thread_sqrt);

            int *h_lattice_1 = (int *) calloc(size, sizeof(int));
            int *h_lattice_2 = (int *) calloc(size, sizeof(int));
            int *lattice_k = (int *) calloc(size, sizeof(int));

           int *d_read;
           int *d_write;

           const int d_lattice_size = size * sizeof(int);

           hipMalloc((void **) &d_read, d_lattice_size);
           hipMalloc((void **) &d_write, d_lattice_size);


           srand((unsigned int) time(NULL));

           for(int iterations=0; iterations < 10; ++iterations) {

               for (int i = 0; i < size; ++i) {
                   int value = (int) rand() % 2;

                   h_lattice_1[i] = value ? 1 : -1;
               }

               hipMemcpy(d_read, h_lattice_1, d_lattice_size, hipMemcpyHostToDevice);
               hipMemcpy(d_write, h_lattice_2, d_lattice_size, hipMemcpyHostToDevice);

               /*** print ***/
               /*
               for(int i=0; i < n; ++i){
                   for(int j=0; j < n; ++j){
                       printf("%d\t", h_lattice_1[i*n + j]);
                   }
                   printf("\n");
               }
               */


               int *temp = NULL;

               /*** CALCUALTE ***/

               dim3 dimBlock(threads_per_block_sqrt, threads_per_block_sqrt);
               dim3 dimGrid(num_blocks_sqrt, num_blocks_sqrt);


               gettimeofday(&start, NULL); //Start timing the computation

               for (int i = 0; i < k; ++i) {

                   v_2<<< dimGrid, dimBlock >>>(d_read, d_write, n, moments_per_thread_sqrt);

                   hipDeviceSynchronize();

                   temp = d_write;
                   d_write = d_read;
                   d_read = temp;

               }

               gettimeofday(&end, NULL); //Stop timing the computation

               myTime = (end.tv_sec + (double) end.tv_usec / 1000000) -
                        (start.tv_sec + (double) start.tv_usec / 1000000);

               hipMemcpy(lattice_k, temp, d_lattice_size, hipMemcpyDeviceToHost);


               /*** print ***/

               printf("%lf\n", myTime);
               /*
               printf("After %d generations:\n", k);

               for(int i=0; i < n; ++i){
                   for(int j=0; j < n; ++j){
                       printf("%d\t", lattice_k[i*n + j]);
                   }
                   printf("\n");
               }
               */


           }

            hipFree(d_write);
            hipFree(d_read);


            free(h_lattice_1);
            free(h_lattice_2);
            free(lattice_k);

        }
    }

    return 0;
}

__global__ void v_2(int* read, int* write, int n, int moments_per_thread_sqrt) {

    int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_id_y = blockIdx.y * blockDim.y + threadIdx.y;

    int thread_i = thread_id_y * moments_per_thread_sqrt;
    int thread_j = thread_id_x * moments_per_thread_sqrt;

    for(int i = thread_i; i < thread_i + moments_per_thread_sqrt; ++i) {
        for(int j = thread_j; j < thread_j + moments_per_thread_sqrt; ++j) {

            int i_minus_one = (n + ((i - 1) % n)) % n;
            int i_plus_one = (n + ((i + 1) % n)) % n;
            int j_minus_one = (n + ((j - 1) % n)) % n;
            int j_plus_one = (n + ((j + 1) % n)) % n;

            int value = (read[i * n + j]
                         + read[i_minus_one * n + j]
                         + read[i_plus_one * n + j]
                         + read[i * n + j_minus_one]
                         + read[i * n + j_plus_one]);

            write[i * n + j] = (value > 0) ? 1 : -1;

        }
    }



}